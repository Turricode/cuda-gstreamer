#include "hip/hip_runtime.h"
#include "library.cuh"

#include <stdio.h>

#define BOX_WIDTH 32
#define BOX_HEIGHT 32

void preprocess(void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat *sformat,
                unsigned int nsurfcount,
                void ** userPtr)
{
//    printf("PREPROCESSING\n");
}

void post_process(void **sBaseAddr,
                  unsigned int *smemsize,
                  unsigned int *swidth,
                  unsigned int *sheight,
                  unsigned int *spitch,
                  ColorFormat *sformat,
                  unsigned int nsurfcount,
                  void ** userPtr)
{
//    printf("POSTPROCESSING\n");
}


__global__ void addLabelsKernel(int* pDevPtr, int pitch){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    char * pElement = (char*)pDevPtr + row * pitch + col * 2;
    pElement[0] = 0;
    pElement[1] = 0;
    return;
}

static int add_labels(hipDevice_t *p_dev_ptr, int pitch){

    dim3 threadsPerBlock(BOX_WIDTH, BOX_HEIGHT);
    dim3 blocks(3,3);
    addLabelsKernel<<<blocks,threadsPerBlock>>>((int*)p_dev_ptr, pitch);
    return 0;

}

void gpu_process(EGLImageKHR image, void ** userPtr){

    hipError_t status;
    CUeglFrame eglFrame;
    hipGraphicsResource_t pResource = NULL;

    hipFree(0);

    status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);

    if (status != hipSuccess) {
        printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
        return;
    }

    status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
    if (status != hipSuccess) {
        printf ("hipGraphicsSubResourceGetMappedArray failed\n");
    }

    status = hipCtxSynchronize();
    if (status != hipSuccess) {
        printf ("hipCtxSynchronize failed \n");
    }

//    if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
//        if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_RGBA) {
//            printf("USING RGBA\n");
//        } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
//            printf("USING YUV420\n");
//
//        } else
//            printf ("Invalid eglcolorformat\n");
//    }
    add_labels((hipDevice_t *) eglFrame.frame.pPitch[0], eglFrame.pitch);

    status = hipCtxSynchronize();
    status = hipGraphicsUnregisterResource(pResource);


    //    printf("GPU_PROCESSING\n");
}

extern "C" void init(CustomerFunction *cf){
    cf->fPreProcess = preprocess;
    cf->fGPUProcess = gpu_process;
    cf->fPostProcess = post_process;
}

extern "C" void deinit(){
    printf("deinited\n");
}