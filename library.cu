#include "hip/hip_runtime.h"
#include "library.cuh"

#include <stdio.h>

/*
 *  total_width = 1920 / 2 = 960
 *  total_height = 1080 / 2 = 540
 *
 *  block_x_count = 960 / 30 = 32
 *  block_y_count = 540 / 30 = 18
 * */

#define BOX_WIDTH 30
#define BOX_HEIGHT 30

void preprocess(void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat *sformat,
                unsigned int nsurfcount,
                void ** userPtr)
{
//    printf("PREPROCESSING\n");
}

void post_process(void **sBaseAddr,
                  unsigned int *smemsize,
                  unsigned int *swidth,
                  unsigned int *sheight,
                  unsigned int *spitch,
                  ColorFormat *sformat,
                  unsigned int nsurfcount,
                  void ** userPtr)
{
//    printf("POSTPROCESSING\n");
}


__global__ void addLabelsKernel(int* pDevPtr, int pitch){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    char * pElement = (char*)pDevPtr + row * pitch + col * 2;
    pElement[0] = (char)abs(0xff - pElement[0]);
    pElement[1] = (char)abs(0xff - pElement[1]);
    return;
}

static int add_labels(hipDevice_t *p_dev_ptr, int pitch){

    dim3 threadsPerBlock(BOX_WIDTH, BOX_HEIGHT);
    dim3 blocks(32,18);
    addLabelsKernel<<<blocks,threadsPerBlock>>>((int*)p_dev_ptr, pitch);
    return 0;

}

void gpu_process(EGLImageKHR image, void ** userPtr){

    hipError_t status;
    CUeglFrame eglFrame;
    hipGraphicsResource_t pResource = NULL;

    hipFree(0);

    status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);

    if (status != hipSuccess) {
        printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
        return;
    }

    status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
    if (status != hipSuccess) {
        printf ("hipGraphicsSubResourceGetMappedArray failed\n");
    }

    status = hipCtxSynchronize();
    if (status != hipSuccess) {
        printf ("hipCtxSynchronize failed \n");
    }

//    if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
//        if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_RGBA) {
//            printf("USING RGBA\n");
//        } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
//            printf("USING YUV420\n");
//
//        } else
//            printf ("Invalid eglcolorformat\n");
//    }
    add_labels((hipDevice_t *) eglFrame.frame.pPitch[1], eglFrame.pitch);

    status = hipCtxSynchronize();
    status = hipGraphicsUnregisterResource(pResource);


    //    printf("GPU_PROCESSING\n");
}

extern "C" void init(CustomerFunction *cf){
    cf->fPreProcess = preprocess;
    cf->fGPUProcess = gpu_process;
    cf->fPostProcess = post_process;
}

extern "C" void deinit(){
    printf("deinited\n");
}